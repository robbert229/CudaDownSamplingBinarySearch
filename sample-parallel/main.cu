
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define ARRAY_LENGTH (100000)
#define ARRAY_ELEMENT_SIZE (sizeof(long))
#define ARRAY_SIZE (ARRAY_ELEMENT_SIZE * ARRAY_LENGTH)
#define SAMPLE_LENGTH (100)
#define SAMPLE_SIZE (ARRAY_ELEMENT_SIZE * SAMPLE_LENGTH)
#define QUERY_LENGTH (200)
#define QUERY_SIZE (ARRAY_ELEMENT_SIZE * QUERY_LENGTH)
#define PRINT_ARRAY (0)

/*
First create a downsampled array (create an array and populate it with every Nth element from array)
Search the downsampled array for the number closest but smaller than desired. Then look in the area between that number and next number in the sample
*/

int random(int min, int max){
	return (rand()%(max-min))+min;	
}

void populateArray(long* array, long i){
	array[i] = i;
	if(PRINT_ARRAY)
		printf("array[%ld] = %ld\n",i,array[i]);
}

__global__ void populateSample(long* array, long *sample){
	long i = threadIdx.x;
	sample[i] = array[i * ARRAY_LENGTH / SAMPLE_LENGTH];
}

void populateQuery(long* query, long i){
	query[i] = random(0,100);
	if(PRINT_ARRAY)
		printf("query[%ld] = %ld\n",i,query[i]);
}

__device__ long binary_search_guess(long *array, long number_of_elements, long key)
{
	long low = 0, high = number_of_elements-1, mid;
	while(low <= high)
	{
		mid = (low + high)/2;
		if(array[mid] < key)
		{
			low = mid + 1; 
		}
		else if(array[mid] == key)
		{
			return mid;
		}
		else if(array[mid] > key)
		{
			high = mid-1;
		}
	}

	if(array[mid] > key){
		while(mid > 0 && array[mid] > key){
			mid--;
		}
	}
	return mid;
}

__device__ long binary_search_precise(long *array, long key,long low, long high)
{
	long mid;
	while(low <= high)
	{
		mid = (low + high)/2;
		if(array[mid] < key)
		{
			low = mid + 1; 
		}
		else if(array[mid] == key)
		{
			return mid;
		}
		else if(array[mid] > key)
		{
			high = mid-1;
		}
	}
	return -1;
}


__global__ void search(long* array, long *sample, long*output, long* query){
	long index = threadIdx.x;
	if(query[index] < *array || query[index] >= sample[SAMPLE_LENGTH-1] + ARRAY_LENGTH / SAMPLE_LENGTH)
		output[index] = -1;
	
	long guess = binary_search_guess(sample,SAMPLE_LENGTH,query[index]);	
	output[index] = binary_search_precise(
		array,
		query[index],
		guess * ARRAY_LENGTH / SAMPLE_LENGTH,
		(guess + 1) * ARRAY_LENGTH / SAMPLE_LENGTH
	); 
}

int main(int argc,char* argv[]){
	long *array = (long*)malloc(ARRAY_SIZE);
	//long *sample = (long*)malloc(SAMPLE_SIZE);
	long *output = (long*)malloc(QUERY_SIZE);
	long *query = (long*)malloc(QUERY_SIZE);

	long *device_array;
	long *device_sample;
	long *device_output;
	long *device_query;

	hipMalloc((void**)&device_array,ARRAY_SIZE);
	hipMalloc((void**)&device_sample,SAMPLE_SIZE);
	hipMalloc((void**)&device_output,QUERY_SIZE);
	hipMalloc((void**)&device_query,QUERY_SIZE);

	// cpu
	printf("Populating Array\n");
	for(long i=0;i<ARRAY_LENGTH;i++)
		populateArray(array,i);

	// cpu
	printf("Populating Query\n");
	for(long i=0;i<QUERY_LENGTH;i++)
		populateQuery(query,i);

	// gpu
	printf("Populating Sample\n");
	populateSample<<<1,SAMPLE_LENGTH>>>(device_array,device_sample);

	hipMemcpy(device_array,array,ARRAY_SIZE,hipMemcpyHostToDevice);
	hipMemcpy(device_query,query,QUERY_SIZE,hipMemcpyHostToDevice);

	// gpu
	printf("Processing Query\n");
	search<<<1,QUERY_LENGTH>>>(device_array,device_sample,device_output,device_query);

	// copy the results from the gpu to the cpu
	hipMemcpy(output,device_output,QUERY_SIZE,hipMemcpyDeviceToHost);
	
	printf("Printing Results\n");
	for(long i=0;i<QUERY_LENGTH;i++)
		printf("results[%ld] = (%ld == [%ld])\n",i,output[i],query[i]);

	hipFree(device_array);
	hipFree(device_sample);
	hipFree(device_output);
	hipFree(device_query);

	free(array);
	free(output);
	free(query);

	return 0;
}